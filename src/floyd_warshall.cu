#include "hip/hip_runtime.h"
#include <iostream> // cout

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "floyd_warshall.h"

__global__ void kernel() {

}

__host__ void floyd_warshall_blocked_cuda() {
  std::cout << "We reached this far!\n";

  kernel<<<1,1>>>();

  // from assignment 1
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  
  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);

    std::cout << "Device " << i << ": " << deviceProps.name << "\n"
	      << "\tSMs: " << deviceProps.multiProcessorCount << "\n"
	      << "\tGlobal mem: " << static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024 * 1024) << "GB \n"
	      << "\tCUDA Cap: " << deviceProps.major << "." << deviceProps.minor << "\n";
  }
}
